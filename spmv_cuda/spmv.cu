#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "util.h"
#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include "anonymouslib_cuda.h"
#include "mmio.h"
using namespace std;

#ifndef VALUE_TYPE
#define VALUE_TYPE double
#endif

#ifndef NUM_RUN
#define NUM_RUN 1000
#endif


	template <typename T>
inline std::string to_string(T value)
{
	std::ostringstream os ;
	os << value ;
	return os.str() ;
}

int call_anonymouslib(int m, int n, int nnzA,
                  int *csrRowPtrA, int *csrColIdxA, VALUE_TYPE *csrValA,
                  VALUE_TYPE *x, VALUE_TYPE *y, VALUE_TYPE alpha)
{
    int err = 0;
    hipError_t err_cuda = hipSuccess;

    // set device
    int device_id = 0;
    hipSetDevice(device_id);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device_id);

    cout << "Device [" <<  device_id << "] " << deviceProp.name << ", " << " @ " << deviceProp.clockRate * 1e-3f << "MHz. " << endl;

    double gb = getB<int, VALUE_TYPE>(m, nnzA);
    double gflop = getFLOP<int>(nnzA);

    // Define pointers of matrix A, vector x and y
    int *d_csrRowPtrA;
    int *d_csrColIdxA;
    VALUE_TYPE *d_csrValA;
    VALUE_TYPE *d_x;
    VALUE_TYPE *d_y;

    // Matrix A
    checkCudaErrors(hipMalloc((void **)&d_csrRowPtrA, (m+1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_csrColIdxA, nnzA  * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_csrValA,    nnzA  * sizeof(VALUE_TYPE)));

    checkCudaErrors(hipMemcpy(d_csrRowPtrA, csrRowPtrA, (m+1) * sizeof(int),   hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_csrColIdxA, csrColIdxA, nnzA  * sizeof(int),   hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_csrValA,    csrValA,    nnzA  * sizeof(VALUE_TYPE),   hipMemcpyHostToDevice));

    // Vector x
    checkCudaErrors(hipMalloc((void **)&d_x, n * sizeof(VALUE_TYPE)));
    checkCudaErrors(hipMemcpy(d_x, x, n * sizeof(VALUE_TYPE), hipMemcpyHostToDevice));

    // Vector y
    checkCudaErrors(hipMalloc((void **)&d_y, m  * sizeof(VALUE_TYPE)));
    checkCudaErrors(hipMemset(d_y, 0, m * sizeof(VALUE_TYPE)));

    anonymouslibHandle<int, unsigned int, VALUE_TYPE> A(m, n);
    err = A.inputCSR(nnzA, d_csrRowPtrA, d_csrColIdxA, d_csrValA);
    //cout << "inputCSR err = " << err << endl;

    err = A.setX(d_x); // you only need to do it once!
    //cout << "setX err = " << err << endl;

    A.setSigma(ANONYMOUSLIB_AUTO_TUNED_SIGMA);

    // warmup device
    A.warmup();

    anonymouslib_timer asCSR5_timer;
    asCSR5_timer.start();

    err = A.asCSR5();

    cout << "CSR->CSR5 time = " << asCSR5_timer.stop() << " ms." << endl;
    //cout << "asCSR5 err = " << err << endl;

    // check correctness by running 1 time
    err = A.spmv(alpha, d_y);
    //cout << "spmv err = " << err << endl;
    checkCudaErrors(hipMemcpy(y, d_y, m * sizeof(VALUE_TYPE), hipMemcpyDeviceToHost));

    // warm up by running 50 times
    if (NUM_RUN)
    {
        for (int i = 0; i < 50; i++)
            err = A.spmv(alpha, d_y);
    }

    err_cuda = hipDeviceSynchronize();

    anonymouslib_timer CSR5Spmv_timer;
    CSR5Spmv_timer.start();

    // time spmv by running NUM_RUN times
    for (int i = 0; i < NUM_RUN; i++)
        err = A.spmv(alpha, d_y);
    err_cuda = hipDeviceSynchronize();

    double CSR5Spmv_time = CSR5Spmv_timer.stop() / (double)NUM_RUN;

    if (NUM_RUN)
        cout << "CSR5-based SpMV time = " << CSR5Spmv_time
             << " ms. Bandwidth = " << gb/(1.0e+6 * CSR5Spmv_time)
             << " GB/s. GFlops = " << gflop/(1.0e+6 * CSR5Spmv_time)  << " GFlops." << endl;

    A.destroy();

    checkCudaErrors(hipFree(d_csrRowPtrA));
    checkCudaErrors(hipFree(d_csrColIdxA));
    checkCudaErrors(hipFree(d_csrValA));
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));

    return err;
}

int main(int argc, char** argv){
	char input_file_name[1024];
	char filename[1024] = {"./kcode/ccd01.cl"};
	parameter param = parse_command_line(argc, argv, input_file_name, NULL, filename);
	// reading rating matrix
	smat_t R;	// val: csc, val_t: csr
	load(input_file_name, R, false, false);
	unsigned int m = R.rows;
	unsigned int n = R.cols;
	unsigned int *ptr = R.row_ptr;
	unsigned int *idx = R.col_idx;
	unsigned int nnz = R.nnz;
	VALUE_TYPE * value = R.val_t;
	cout << " ( " << m << ", " << n << " ) nnz = " << nnz << endl;

	// native spmv
	VALUE_TYPE *x = (VALUE_TYPE *)malloc(n * sizeof(VALUE_TYPE));
	for(unsigned int i = 0; i < n; i++)
		x[i] = (VALUE_TYPE)i * 0.1;
	VALUE_TYPE *y = (VALUE_TYPE *)malloc(m * sizeof(VALUE_TYPE));
	VALUE_TYPE *y_ref = (VALUE_TYPE *)malloc(m * sizeof(VALUE_TYPE));

	// for debugging
	srand(time(NULL));
	for(unsigned int i=0; i<nnz; i++)
	{
		value[i] = rand() % 10;
	}

	// compute reference results on a cpu core
	VALUE_TYPE alpha = 1.0;
	double gb = getB<int, VALUE_TYPE>(m, nnz);
	double gflop = getFLOP<int>(nnz);
	double t = gettime();
	int ref_iter = 1;
	for (int iter = 0; iter < ref_iter; iter++){
		for (int i = 0; i < m; i++){
			VALUE_TYPE sum = 0;
			for (int j = ptr[i]; j < ptr[i+1]; j++)
				sum += x[idx[j]] * value[j] * alpha;
			y_ref[i] = sum;
		}
	}

	double ref_time = (gettime() - t) / (double)ref_iter;
	cout << "cpu sequential time = " << ref_time
		<< " ms. Bandwidth = " << gb/(1.0e+6 * ref_time)
		<< " GB/s. GFlops = " << gflop/(1.0e+6 * ref_time)  << " GFlops." << endl << endl;
	// compute spmv on the ocl device
	call_anonymouslib(m, n, nnz, (int *)ptr, (int *)idx, value, x, y, alpha);

	// compare reference and anonymouslib results
	int error_count = 0;
	for (int i = 0; i < m; i++)
		if (abs(y_ref[i] - y[i]) > 0.01 * abs(y_ref[i]))
		{
			error_count++;
			cout << y_ref[i] << "," << y[i] << "\t";
		}
	if (error_count == 0)
		cout << "Check... PASS!" << endl;
	else
		cout << "Check... NO PASS! #Error = " << error_count << " out of " << m << " entries." << endl;
	cout << "------------------------------------------------------" << endl;

	if(x!=NULL) free(x);
	if(y!=NULL) free(y);
	if(y_ref!=NULL) free(y_ref);
	return 0;
}


